
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
if (comp > cosf(+1.6313E-42f)) {
  float tmp_1 = (var_1 * var_2 * logf(+1.5851E35f));
float tmp_2 = +1.5440E-22f;
comp += tmp_2 / tmp_1 + coshf(-1.1385E-20f - +0.0f - var_3);
comp = (var_4 + atanf((var_5 / (var_6 / (+1.8340E0f + var_7)))));
if (comp >= var_8 * (var_9 - -1.2554E-42f)) {
  comp = (-0.0f / -1.0205E-43f * var_10 - +1.1519E-35f);
float tmp_3 = (var_11 * (var_12 * var_13));
comp = tmp_3 + var_14 + var_15 / var_16;
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
