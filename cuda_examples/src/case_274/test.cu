
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
if (comp < acosf((var_3 + (-1.9863E-35f / -1.3315E36f * (var_4 / var_5 * -1.3237E-35f))))) {
  for (int i=0; i < var_1; ++i) {
    comp = -0.0f / (var_6 * (var_7 - (-1.4789E34f / sqrtf(-1.1994E13f - +1.2658E36f / asinf(sinf(-1.8300E-36f))))));
if (comp == var_8 * +0.0f) {
  comp = (var_9 / (var_10 * +1.5083E-43f));
}
for (int i=0; i < var_2; ++i) {
  comp = (-1.3374E-37f * var_11 * (-0.0f / logf((var_12 + (var_13 / -0.0f + (var_14 + var_15 + var_16))))));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
